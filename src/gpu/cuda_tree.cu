#include "hip/hip_runtime.h"
#include "cuda_tree.hpp"
#include "resources.hpp"
#include "cuda_thrust_ops.hpp"
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include "cuda_util.hpp"

namespace gmp { namespace tree {

    using namespace gmp::thrust_ops;

    template <typename IndexType>
    traverse_result_t<IndexType>::traverse_result_t(size_t max_num_mc_) : 
        num_indexes(0),
        max_num_mc(max_num_mc_) 
    {
        auto stream = gmp::resources::gmp_resource::instance().get_stream();
        auto dm = gmp::resources::gmp_resource::instance().get_device_memory_manager();
        indexes = static_cast<IndexType*>(dm->allocate(max_num_mc_ * sizeof(IndexType), stream));
    }

    template <typename IndexType>
    traverse_result_t<IndexType>::~traverse_result_t()
    {
        auto dm = gmp::resources::gmp_resource::instance().get_device_memory_manager();
        auto stream = gmp::resources::gmp_resource::instance().get_stream();
        if (indexes) dm->deallocate(indexes, stream);
        indexes = nullptr;
    }

    template class traverse_result_t<int32_t>;
    template class traverse_result_t<uint32_t>;

    // binary radix tree implementations
    template <typename MortonCodeType, typename IndexType>
    __global__
    void build_tree_kernel(const MortonCodeType* morton_codes, const IndexType num_mc, const IndexType num_bits, internal_node_t<MortonCodeType, IndexType>* internal_nodes)
    {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid >= num_mc - 1) return;  // We have num_mc - 1 internal nodes

        IndexType first, last;
        morton_codes::determine_range<MortonCodeType, IndexType>(morton_codes, num_mc, tid, first, last, num_bits);
        IndexType delta_node = morton_codes::delta<MortonCodeType, IndexType>(morton_codes, num_mc, first, last, num_bits);
        IndexType split = morton_codes::find_split<MortonCodeType, IndexType>(morton_codes, num_mc, delta_node, first, last, num_bits);
        MortonCodeType lower_bound, upper_bound;
        morton_codes::find_lower_upper_bounds<MortonCodeType, IndexType>(morton_codes[split], delta_node, lower_bound, upper_bound, num_bits);

        // Determine left and right children
        // n is the number of leaf nodes (num_mc)
        IndexType n = num_mc;
        IndexType left = (split == first) ? split : split + n;
        IndexType right = (split + 1 == last) ? split + 1 : split + 1 + n;

        internal_nodes[tid] = internal_node_t<MortonCodeType, IndexType>(left, right, lower_bound, upper_bound);
    }

    template <typename MortonCodeType, typename IndexType>
    cuda_binary_radix_tree_t<MortonCodeType, IndexType> cuda_build_tree(const vector_device<MortonCodeType>& morton_codes, const IndexType num_bits)
    {
        assert(num_bits % 3 == 0);
        using inode_t = internal_node_t<MortonCodeType, IndexType>;

        auto stream = gmp::resources::gmp_resource::instance().get_stream();
        auto dm = gmp::resources::gmp_resource::instance().get_device_memory_manager();

        cuda_binary_radix_tree_t<MortonCodeType, IndexType> tree;
        auto num_mc = static_cast<IndexType>(morton_codes.size());
        tree.internal_nodes = static_cast<inode_t*>(dm->allocate((num_mc - 1) * sizeof(inode_t), stream));
        tree.leaf_nodes = static_cast<MortonCodeType*>(dm->allocate(num_mc * sizeof(MortonCodeType), stream));
        tree.num_leaf_nodes = num_mc;

        dim3 block_size(256, 1, 1), grid_size(1, 1, 1);
        grid_size.x = (num_mc - 1 + block_size.x - 1) / block_size.x;
        build_tree_kernel<MortonCodeType, IndexType><<<grid_size, block_size, 0, stream>>>(morton_codes.data(), num_mc, num_bits, tree.internal_nodes);
        
        // copy leaf nodes
        hipMemcpyAsync(tree.leaf_nodes, morton_codes.data(), num_mc * sizeof(MortonCodeType), hipMemcpyDeviceToDevice, stream);

        bind_texture_memory(tree.internal_nodes, (num_mc - 1) * sizeof(inode_t), 32, tree.internal_nodes_tex);
        bind_texture_memory(tree.leaf_nodes, num_mc * sizeof(MortonCodeType), 32, tree.leaf_nodes_tex);
        return tree;
    }

    template cuda_binary_radix_tree_t<uint32_t, int32_t> cuda_build_tree(const vector_device<uint32_t>& morton_codes, const int32_t num_bits);


    template <typename MortonCodeType, typename IndexType>
    void cuda_binary_radix_tree_t<MortonCodeType, IndexType>::get_internal_nodes(internal_node_t<MortonCodeType, IndexType>* internal_nodes) const
    {
        auto stream = gmp::resources::gmp_resource::instance().get_stream();
        hipMemcpyAsync(internal_nodes, this->internal_nodes, (num_leaf_nodes - 1) * sizeof(inode_t), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        return;
    }

    template <typename MortonCodeType, typename IndexType>
    void cuda_binary_radix_tree_t<MortonCodeType, IndexType>::get_leaf_nodes(MortonCodeType* leaf_nodes) const
    {
        auto stream = gmp::resources::gmp_resource::instance().get_stream();
        hipMemcpyAsync(leaf_nodes, this->leaf_nodes, num_leaf_nodes * sizeof(MortonCodeType), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        return;
    }

    template <typename MortonCodeType, typename IndexType>
    __device__
    void tree_traverse_kernel(hipTextureObject_t internal_nodes_tex, hipTextureObject_t leaf_nodes_tex, const IndexType num_leaf_nodes, 
        const cuda_compare_op_t<MortonCodeType, IndexType>& check_method, const array3d_t<IndexType>& cell_shifts,
        IndexType* indexes, size_t& num_indexes)
    {
        // Fixed stack for traversal
        IndexType stack_data[64];
        int stack_top = -1;
        
        IndexType result_index = 0;
        IndexType n = num_leaf_nodes;
        
        // Start with root (internal node index n)
        stack_data[++stack_top] = n;
        
        while (stack_top >= 0) {
            IndexType node_index = stack_data[stack_top--];
            
            if (node_index < n) {
                // Leaf node
                MortonCodeType morton_code = tex1Dfetch<MortonCodeType>(leaf_nodes_tex, node_index);
                
                // Check if morton code is within query bounds
                check_method(morton_code, cell_shifts, node_index, indexes, num_indexes);
            } else {
                // Internal node
                IndexType left = tex1Dfetch<IndexType>(internal_nodes_tex, (node_index - n) * 4);
                IndexType right = tex1Dfetch<IndexType>(internal_nodes_tex, (node_index - n) * 4 + 1);
                MortonCodeType lower_bound = tex1Dfetch<MortonCodeType>(internal_nodes_tex, (node_index - n) * 4 + 2);
                MortonCodeType upper_bound = tex1Dfetch<MortonCodeType>(internal_nodes_tex, (node_index - n) * 4 + 3);
                
                if (check_method(lower_bound, upper_bound, cell_shifts)) {
                    if (stack_top < 63) stack_data[++stack_top] = left;
                    if (stack_top < 63) stack_data[++stack_top] = right;
                }
            }
        }
        return;
    }
    
    template __device__
    void tree_traverse_kernel<uint32_t, int32_t>(hipTextureObject_t, hipTextureObject_t,
        int32_t, const cuda_compare_op_t<uint32_t, int32_t>&, const array3d_t<int32_t>&, int32_t*, size_t&);

    void bind_texture_memory(void* data_ptr, size_t size, int bits_per_channel, hipTextureObject_t& tex)
    {
        // Create texture descriptor for internal nodes
        hipResourceDesc resDesc_internal = {};
        resDesc_internal.resType = hipResourceTypeLinear;
        resDesc_internal.res.linear.devPtr = data_ptr;
        resDesc_internal.res.linear.desc.f = hipChannelFormatKindUnsigned;
        resDesc_internal.res.linear.desc.x = bits_per_channel; // bits per channel
        resDesc_internal.res.linear.sizeInBytes = size;
        
        hipTextureDesc texDesc = {};
        texDesc.readMode = hipReadModeElementType;
        texDesc.normalizedCoords = 0;
        texDesc.filterMode = hipFilterModePoint;
        texDesc.addressMode[0] = hipAddressModeClamp;
        
        hipCreateTextureObject(&tex, &resDesc_internal, &texDesc, nullptr);
    }

    
    void unbind_texture_memory(hipTextureObject_t tex)
    {
        hipDestroyTextureObject(tex);
    }

    // Explicit instantiations for cuda_binary_radix_tree_t (used in tests)
    template class cuda_binary_radix_tree_t<uint32_t, int32_t>;

}} // namespace gmp::tree 